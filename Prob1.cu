
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void square(float *d_out, float *d_in,float *d_in2){
	int idx = threadIdx.x;
	float f = d_in[idx];
    float g=d_in2[idx];
	d_out[idx] = f+g;
}


int main(){
    const int ARRAY_SIZE = 96;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);
	
	float h_in[ARRAY_SIZE];
    float h_in2[ARRAY_SIZE];
    float h_out[ARRAY_SIZE];
	for (int i=0; i < ARRAY_SIZE; i++){
		h_in[i] = float(i);
        h_in2[i] = float(i) + 3;
        h_out[i] = h_in[i] + h_in2[i];


	}
	//float h_out[ARRAY_SIZE];

	float *d_in;
    float *d_in2;
	float *d_out;

	hipMalloc((void**) &d_in, ARRAY_BYTES); 
    hipMalloc((void**) &d_in2, ARRAY_BYTES); 
	hipMalloc((void**) &d_out, ARRAY_BYTES); 

	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_in2 , h_in2, ARRAY_BYTES, hipMemcpyHostToDevice);

	square<<<1, ARRAY_SIZE>>>(d_out, d_in, d_in2);

	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

//	for (int i =0; i<ARRAY_SIZE; i++){
//		printf("%f", h_out[i]);
//		printf(((i % 4) !=3)? "\t": "\n");
//	}


        printf("Matrix 1 \n");
    for(int craig = 0; craig < 4 ; craig++)
    {
        printf("%f \t" , h_in[craig]);
        if(craig == 1){
            puts("");
        }
    }
    puts("");

    printf("Matrix 2 \n");
    for(int craig = 0; craig < 4 ; craig++)
    {
        printf("%f \t" , h_in2[craig]);
        if(craig == 1){
            puts("");
        }
    }
    puts("");

    printf("------------------------------------------------\n");
    printf("Matric ADDITION\n");
    printf("Left is D1, Middle is D2, right is D out\n");
    for(int j = 0; j < 4; j++)
    {
        printf("%f \t " , h_in[j]); // Array 1
        printf("%f \t ", h_in2[j]); // Array 2
        printf("%f \t", h_out[j]); // Final 
        puts("");
    }
	
    printf("------------------------------------------------\n");


	hipFree(d_in);
    hipFree(d_in2);
	hipFree(d_out);

	return 0;
}